#include "hip/hip_runtime.h"
#include <iostream>        // includes, standard template & armadillo library
#include <armadillo>
//#include <hip/hip_runtime.h>  // includes, cuda's runtime & fft
//#include <hipfft/hipfft.h>
//#include <hipfft/hipfftXt.h>
//#include <hiprand/hiprand_kernel.h> // Device random API
//#include <ctime> // CPU timer
//#include "../inst/include/constant.h"  // math constants
//#include "../inst/include/random.h"
#include "../inst/include/density.h"  

#include "../inst/include/common.h"
#include "../inst/include/reduce.h"

unsigned int nextPow2(unsigned int x) {
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}


/* -------------------------------------------------------------------------  
 KDE operations 
------------------------------------------------------------------------- */
void summary(int *nsim, unsigned int *d_R, float *d_RT, float *out) {
    unsigned int maxThread = 256;
    unsigned int nThread = (*nsim < maxThread) ? nextPow2(*nsim) : maxThread;
    unsigned int nBlk    = ((*nsim) + nThread ) / nThread / 2;

    float *h_n1min_out, *h_n1max_out, *h_sum_out, *h_sqsum_out;
    float *d_n1min_out, *d_n1max_out, *d_sum_out, *d_sqsum_out;
    unsigned int *h_count_out, *h_nsim;
    unsigned int *d_count_out, *d_nsim;
  
    size_t dBlkfSize = nBlk * sizeof(float) * 2;
    size_t blkfSize  = nBlk * sizeof(float);
    size_t dBlkuSize = nBlk * sizeof(unsigned int) * 2;
    size_t uSize     = 1 * sizeof(unsigned int);
  
    h_nsim      = (unsigned int *)malloc(uSize);
    h_n1min_out = (float *)malloc(blkfSize);
    h_n1max_out = (float *)malloc(blkfSize);
    h_sum_out   = (float *)malloc(blkfSize);
    h_sqsum_out = (float *)malloc(dBlkfSize);
    h_count_out = (unsigned int *)malloc(dBlkuSize);
    // must reset h_count_out back to 0
    for(int i=0; i<2*nBlk; i++) { h_count_out[i] = 0; } 
    *h_nsim = (unsigned int)*nsim;

    CHECK(hipMalloc((void**) &d_nsim,      uSize));
    CHECK(hipMalloc((void**) &d_n1min_out, blkfSize));
    CHECK(hipMalloc((void**) &d_n1max_out, blkfSize));
    CHECK(hipMalloc((void**) &d_sum_out,   blkfSize));
    CHECK(hipMalloc((void**) &d_sqsum_out, dBlkfSize));
    CHECK(hipMalloc((void**) &d_count_out, dBlkuSize));
  
    CHECK(hipMemcpy(d_nsim,      h_nsim,  uSize,  hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_count_out, h_count_out, dBlkuSize, hipMemcpyHostToDevice));

    // must be first min and then max
    count_kernel<<<2*nBlk, nThread>>>(d_nsim, d_R, d_count_out); hipFree(d_R);
    n1min_kernel<<<nBlk, nThread>>>(d_RT, d_n1min_out); 
    n1max_kernel<<<nBlk, nThread>>>(d_RT, d_n1max_out);
    sum_kernel<<<nBlk, nThread>>>(d_RT,   d_sum_out);
    squareSum_kernel<<<2*nBlk, nThread>>>(d_nsim, d_RT, d_sqsum_out);
  
    CHECK(hipMemcpy(h_n1min_out, d_n1min_out, blkfSize,  hipMemcpyDeviceToHost)); hipFree(d_n1min_out);
    CHECK(hipMemcpy(h_n1max_out, d_n1max_out, blkfSize,  hipMemcpyDeviceToHost)); hipFree(d_n1max_out);
    CHECK(hipMemcpy(h_sum_out,   d_sum_out,   blkfSize,  hipMemcpyDeviceToHost)); hipFree(d_sum_out);
    CHECK(hipMemcpy(h_sqsum_out, d_sqsum_out, dBlkfSize, hipMemcpyDeviceToHost)); hipFree(d_sqsum_out);
    CHECK(hipMemcpy(h_count_out, d_count_out, dBlkuSize, hipMemcpyDeviceToHost)); hipFree(d_count_out);

    arma::vec min_tmp(nBlk); arma::vec max_tmp(nBlk);
    float sum = 0, sqsum = 0;
    for (int i=0; i<2*nBlk; i++) {
      sqsum += h_sqsum_out[i];
      if ( i < nBlk ) {
        min_tmp[i] = (double)h_n1min_out[i];
        max_tmp[i] = (double)h_n1max_out[i];
        sum += h_sum_out[i];
      }
    }

    free(h_sqsum_out); free(h_n1min_out); free(h_n1max_out); free(h_sum_out);
    out[0] = min_tmp.min();
    out[1] = max_tmp.max();
    out[2] = std::sqrt( (sqsum - (sum*sum) / h_count_out[0]) / (h_count_out[0] - 1) );
    out[3] = h_count_out[0]; free(h_count_out);

    // printf("RT0 [minimum maximum]: %.2f %.2f\n", min_tmp.min(), max_tmp.max());
    // printf("RT0 [sum sqsum]: %.2f %.2f\n", sum, sqsum);
    // printf("RT0 [nsRT0 sd]: %f %f\n", out[3], out[2]);
    free(h_nsim); hipFree(d_nsim);
}

void histc(int *nsim, int ngrid, float *h_binedge, float *d_RT, unsigned int *h_hist)
{
    size_t ngrid_plus1fSize = (ngrid + 1) * sizeof(float);
    size_t ngriduSize = ngrid * sizeof(unsigned int);

    float *d_binedge;
    unsigned int *d_hist;
    unsigned int *h_nsim, *d_nsim;
    h_nsim  = (unsigned int *)malloc(sizeof(unsigned int) * 1);
    *h_nsim = (unsigned int)*nsim;
    CHECK(hipMalloc((void**) &d_nsim, sizeof(unsigned int) * 1));
    CHECK(hipMemcpy(d_nsim,   h_nsim, sizeof(unsigned int) * 1,  hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**) &d_binedge, ngrid_plus1fSize)); // 1025
    CHECK(hipMalloc((void**) &d_hist,    ngriduSize));       // 1024
    CHECK(hipMemcpy(d_binedge, h_binedge, ngrid_plus1fSize, hipMemcpyHostToDevice)); free(h_binedge);
    CHECK(hipMemcpy(d_hist,    h_hist,    ngriduSize,       hipMemcpyHostToDevice));
    histc_kernel<<<*nsim/ngrid, ngrid>>>(d_binedge, d_RT, d_nsim, d_hist);
    hipFree(d_RT); hipFree(d_binedge); hipFree(d_nsim);

    CHECK(hipMemcpy(h_hist, d_hist, ngriduSize, hipMemcpyDeviceToHost)); 
    hipFree(d_hist); free(h_nsim); 
}
/*
arma::vec getEdges(arma::vec z, double dt)
{
  arma::vec term1 = z - 0.5*dt;
  arma::vec term2(1) ;
  term2.fill(z[z.size()-1] + 0.5*dt) ;
  return arma::join_cols(term1, term2);
}

arma::vec getFilter(double m, double M, double h, double p) {
  // cannoical Gaussian kernel
  double tmp0    = 2 * arma::datum::pi * (std::pow(2, p) / (M-m)) * 0.5;
  arma::vec tmp1 = arma::linspace<arma::vec>(0, 1, 1 + (std::pow(2, p)/2));
  arma::vec freq = tmp0 * tmp1 ;
  arma::vec s2   = arma::pow(freq, 2) ; // s^2 on p17
  double h2      = h * h;
  arma::vec fil0 = arma::exp(-0.5 * h2 * s2) ;
  arma::vec fil1 = arma::flipud(fil0.rows(1, (fil0.size() - 2)));
  arma::vec out  = arma::join_cols(fil0, fil1) ;
  return out ;
}
*/

arma::vec pmax(arma::vec v, double min)
{
  for (arma::vec::iterator it=v.begin(); it!=v.end(); it++)
  {
    if (*it < min) *it = min ;
  }
  return v ;
}

arma::vec getVec(double *x, int *nx)
{
  arma::vec out(*nx);
  for(int i=0; i<*nx; i++) { out[i]=*(x+i); }
  return out;
}

arma::vec density(arma::vec y, arma::vec be, double dt)
{
  // y is yhat; be is binEdges; ns is nSamples
  arma::uvec hc       = arma::histc(y, be) ;
  arma::vec bincount  = arma::conv_to<arma::vec>::from(hc);
  int ns              = arma::accu(bincount);
  arma::vec PDF_hist  = bincount / (dt * ns);
  arma::vec out       = PDF_hist.rows(0, (PDF_hist.size() - 2)) ;
  return out ;
}

double cquantile(arma::vec y, double q)
{
  arma::vec sy = sort(y);
  int nth = sy.n_elem*(q - (1e-9));
  return sy(nth);
}

double bwNRD0(arma::vec y, double m)
{ // y must be a simulation vector
  // double h   = (q75-q25)/1.34 ; // R divides 1.34
  int n = y.n_elem ;
  return m*0.9*std::min((cquantile(y, 0.75) - cquantile(y, 0.25)),
                      arma::stddev(y))*std::pow((double)n, -0.2);
}

double gaussian(double y, arma::vec yhat, double h) {
  // standard gaussian kernel mean=0; sigma==1
  double x;
  int ns = yhat.n_elem;
  arma::vec result(ns);
  for(arma::vec::iterator it=yhat.begin(); it!=yhat.end(); ++it)
  {
    int i = std::distance(yhat.begin(), it);
    x = (y - *it)/h;  // z / h
    // (1/h) * K(z/h); K_h(z)
    result[i] = ( (1/(sqrt(2*arma::datum::pi))) * exp( -pow(x,2) / 2 ) ) / h;
  }
  // (1/N_s) * sigma K_h (x-x.tidle_j)
  return ( arma::sum(result) / ns);
}

